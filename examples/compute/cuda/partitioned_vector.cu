//  Copyright (c) 2016 Hartmut Kaiser
//
//  Distributed under the Boost Software License, Version 1.0. (See accompanying
//  file LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#define HPX_APPLICATION_NAME partitioned_vector_cu
#define HPX_APPLICATION_STRING "partitioned_vector_cu"
#define HPX_APPLICATION_EXPORTS

#include <hpx/include/compute.hpp>
#include <hpx/include/partitioned_vector.hpp>

#include <hpx/hpx_init.hpp>

#include <iostream>
#include <numeric>
#include <string>
#include <vector>

///////////////////////////////////////////////////////////////////////////////
// Define the partitioned vector types to be used.
typedef hpx::compute::cuda::allocator<int> target_allocator;
typedef hpx::compute::vector<int, target_allocator> target_vector;

HPX_REGISTER_PARTITIONED_VECTOR(int, target_vector);

///////////////////////////////////////////////////////////////////////////////
int hpx_main(boost::program_options::variables_map& vm)
{
    unsigned int seed = (unsigned int)std::time(nullptr);
    if (vm.count("seed"))
        seed = vm["seed"].as<unsigned int>();

    std::cout << "using seed: " << seed << std::endl;
    std::srand(seed);

    // TODO: add more

    return hpx::finalize();
}

int main(int argc, char* argv[])
{
    // add command line option which controls the random number generator seed
    using namespace boost::program_options;
    options_description desc_commandline(
        "Usage: " HPX_APPLICATION_STRING " [options]");

    desc_commandline.add_options()
        ("seed,s", value<unsigned int>(),
        "the random number generator seed to use for this run")
        ;

    // Initialize and run HPX
    return hpx::init(desc_commandline, argc, argv);
}
