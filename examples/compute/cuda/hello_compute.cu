///////////////////////////////////////////////////////////////////////////////
//  Copyright (c) 2016 Thomas Heller
//
//  Distributed under the Boost Software License, Version 1.0. (See accompanying
//  file LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)
///////////////////////////////////////////////////////////////////////////////

#include <hpx/compute/cuda.hpp>
#include <hpx/compute/vector.hpp>

#include <hpx/hpx_init.hpp>

#include <numeric>
#include <iostream>
#include <vector>

void cuda_main(unsigned int seed)
{
    hpx::compute::cuda::target target;

    int const N = 100;
    std::vector<int> h_A(N);
    std::vector<int> h_B(N);
    std::vector<int> h_C_ref(N);
    std::vector<int> h_C(N);

    std::iota(h_A.begin(), h_A.end(), (std::rand() % 100) + 2);
    std::iota(h_B.begin(), h_B.end(), (std::rand() % 100) + 2);

    std::transform(h_A.begin(), h_A.end(), h_B.begin(), h_C_ref.begin(),
        [](int a, int b) { return a + b; });

    {
        hpx::compute::cuda::detail::scoped_active_target active(target);
        int *d_A = 0;
        int *d_B = 0;
        int *d_C = 0;

        hipMalloc(&d_A, N * sizeof(int));
        hipMalloc(&d_B, N * sizeof(int));
        hipMalloc(&d_C, N * sizeof(int));

        hipMemcpy(d_A, h_A.data(), N * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B.data(), N * sizeof(int), hipMemcpyHostToDevice);

        int threadsPerBlock = 256;
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

        hpx::compute::cuda::detail::launch(target, blocksPerGrid, threadsPerBlock,
            [=] __device__ ()
            {
                int i = blockDim.x * blockIdx.x + threadIdx.x;
                if(i < N)
                    d_C[i] = d_A[i] + d_B[i];
            });

        hipMemcpy(h_C.data(), d_C, N * sizeof(int), hipMemcpyDeviceToHost);
        hipStreamSynchronize(active.stream());
#if !defined(__CUDA_ARCH__)
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
#endif
    }

    for(int i = 0; i < N; ++i)
    {
        if(h_C[i] != h_C_ref[i] || h_C[i] != h_A[i] + h_B[i])
        {
            std::cout << "Error at " << i << "\n";
        }
    }
}

int hpx_main(boost::program_options::variables_map& vm)
{
    unsigned int seed = (unsigned int)std::time(0);
    if (vm.count("seed"))
        seed = vm["seed"].as<unsigned int>();

    std::cout << "using seed: " << seed << std::endl;
    std::srand(seed);

    cuda_main(seed);

    return hpx::finalize();
}

#include <string>

int main(int argc, char* argv[])
{
    // add command line option which controls the random number generator seed
    using namespace boost::program_options;
    options_description desc_commandline(
        "Usage: " HPX_APPLICATION_STRING " [options]");

    desc_commandline.add_options()
        ("seed,s", value<unsigned int>(),
        "the random number generator seed to use for this run")
        ;

    // Initialize and run HPX
    return hpx::init(desc_commandline, argc, argv);
}
